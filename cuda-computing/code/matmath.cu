#include "hip/hip_runtime.h"
/**
 * Contains utility functions for performing simple operations on matrices.
 *
 * @authors: Elias Gabriel, Colin Snow
 **/
#include "matmath.h"

/*
  The GPU kernel (or thread) responsible for calculating a single
	value in the output matrix c. This is replicated for the block grid
  running on the GPU to complete the entire calculation.
 */
__global__ void multkernel(Matrix* a, Matrix* b, Matrix* c) {
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
  // holds the intermediate matrix product
	float psum = 0.0;

	// during thread and grid creation it is likely that we needed
  // to round up the number of blocks to a whole number to fit our
  // problemspace. in ceiling the number of blocks, the GPU will
  // execute more threads than we actually need. due to those excess
  // threads, we must ensure that the thread we are currently running
  // is actually within the bounds of our problem and not a throw-away
  //
  // translation: make sure the current thread is actually doing work
	if(row < a->rows && col < b->cols) {
	  // iterate through the row and multiply each element then add it to the total
    for(unsigned int i = 0; i < a->cols; ++i) {
      psum += (a->elements[row*b->cols + i]) * (b->elements[i*b->cols + row]);
	  }
  }

  // store the partial sum in the output matrix
	c->elements[c->cols*row + col] = psum;
}

/*
  Allocates a new block of Unified Memory (URAM) and assigns it to store the
	elements for the provided matrix. URAM is accessible from the CPU and GPU
  and removes the necessity for structure-value copying.
 */
int matmalloc(Matrix** pmat, unsigned int rows, unsigned int cols) {
	hipError_t err = hipMallocManaged(pmat, sizeof **pmat);
	if(err != hipSuccess) {
		fprintf(stderr, "Allocated matrix in URAM: %s\n", hipGetErrorString(err));
		return EXIT_FAILURE_ALLOC;
	}

  Matrix* mat = *pmat;
  mat->rows = rows;
	mat->cols = cols;

  err = hipMallocManaged(&mat->elements, rows * cols * sizeof(float));
  if(err != hipSuccess) {
		fprintf(stderr, "Allocated inner matrix in URAM: %s\n", hipGetErrorString(err));
		return EXIT_FAILURE_ALLOC;
	}
	
  return EXIT_SUCCESS;
}

/*
  Releases a previously-allocated matrix and it's elements from Unified
  Memory. Future CPU/GPU manipulations to M will SEGFAULT.
 */
int matfree(Matrix* M) { 
  hipError_t err = hipFree(M->elements);
  if(err != hipSuccess || (hipFree(M) != hipSuccess)) {
    fprintf(stderr, "Freed matrix from URAM: %s\n", hipGetErrorString(err));
    return EXIT_FAILURE_FREE;
  }

  return EXIT_SUCCESS;
}

/*
  Multiplies the given matrices A and B. A matrix of sufficient size is
  allocated in Unified Memory (URAM) to store the matrix product and is
  assigned to the pointer chain C.

  NOTES: Reference links to better understand CUDA's execution schema
    - https://stackoverflow.com/a/2392271
    - https://devblogs.nvidia.com/even-easier-introduction-cuda/
    - https://devblogs.nvidia.com/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
    - https://stackoverflow.com/a/33247118
 */
int matmultiply(Matrix* A, Matrix* B, Matrix** C) {
  // the GPU must spawn threads in multiples of the warp size, which is
  // 32 on all chips to date, so its more efficient to allocate threads
  // of that factor
  //
  // TODO: scale tpb by matrix ratio but maintain multiple of 32 between 128-512
  dim3 threadsPerBlock(16, 16); // lowest square multiple of 32

  // find the minimum number of blocks to fit the data, overfitting if we cannot
  // roundly divide the number of threads to fit our matrix dimensions
  dim3 numBlocks(((*C)->cols + threadsPerBlock.x - 1) / threadsPerBlock.x, ((*C)->rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

	// Run the kernel with the set sizes and the variables a,b,c
	multkernel<<<numBlocks, threadsPerBlock>>>(A, B, *C);

	// wait to synchronize and join all the running CUDA cores
  // this ensures the calculation is completed
	hipError_t cerr = hipDeviceSynchronize();
	if(cerr != hipSuccess) {
		printf("Synchronizing GPU threads: %s\n", hipGetErrorString(cerr));
		return EXIT_FAILURE_SYNC;
	}

	return EXIT_SUCCESS;
}

/*
   Converts the given matrix to a string represenation and prints
   it to the standard output.
*/
void matprint(Matrix* M) {
  printf("[ ");

  for(unsigned int i = 0; i < M->rows; ++i) {
    for(unsigned int j = 0; j < M->cols; ++j) {
      if(i + j > 0) printf("  ");
      printf("%g ", M->elements[M->cols*i + j]);
    }

    if(i < M->rows - 1) printf("\n");
  }

  printf("]\n");
}
